#include "hip/hip_runtime.h"
//============================================================================
// Name        : main.cu
// Author      : Matt Angus
// Version     : 1.0.0
// Description : Entry point
//============================================================================

#include <exception>
#include <ctime>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "run.cuh"
#include <experimental/filesystem>
#include <regex>
#include <thread>
#include <chrono>
#include "helpers.h"
#include <opencv2/opencv.hpp>

namespace fs = experimental::filesystem;

/** 
 * custom exceptions
 */
class bad_argument: public exception
{
	virtual const char* what() const throw()
	{
		return "Please supply the correct parameters";
	}
} ba_ex;

class bad_image: public exception
{
	virtual const char* what() const throw()
	{
		return "The image at the path supplied coult not be opened";
	}
} bi_ex;

bool hasEnding (std::string const &fullString, std::string const &ending) {
    if (fullString.length() >= ending.length()) {
        return (0 == fullString.compare (fullString.length() - ending.length(), ending.length(), ending));
    } else {
        return false;
    }
}

std::vector<std::string> GetImagesToProcess(std::string& inputPath, std::string& outputPath)
{
	std::vector<std::string> ret;
	for(auto& p: fs::recursive_directory_iterator(inputPath))
	{
		std::string curPath = p.path().string();
		bool regFile = fs::is_regular_file(p);
		if(regFile && hasEnding(curPath, "png"))
		{
			std::string imgPath = std::regex_replace(curPath, std::regex(inputPath), outputPath);

			if(!fs::is_regular_file(imgPath))
			{
				ret.push_back(curPath);
			}
		}
	}
	return ret;
}

void processImage(string image_path, string output_path, string depth_path, float cutOff, cv::Mat kernel)
{
	// output program input state
	std::cout << "reading from path " << image_path << "..." << endl;
	std::cout << "output path " << output_path << "..." << endl;
	//std::cout << "depth path " << depth_path << "..." << std::endl;
	
	// read image data in bgr then copy to standard array
	cv::Mat im;
	while(!im.data)
	{
		im = imread(image_path, CV_LOAD_IMAGE_COLOR);
		if(!im.data) //only be able to parse if IEND chunk is found (i.e. transer complete)
			std::this_thread::sleep_for(std::chrono::milliseconds(100));
	}

	vector<float> depthData;
	{//read depth
		std::ifstream depthFile(depth_path.c_str(), std::ios::binary | std::ios::in);
		
		if(depthFile.good())
		{
			while (!depthFile.eof())
			{
				float temp;
				depthFile.read((char*)&temp, sizeof(temp));
				depthData.push_back(temp);
			}
		}
		else
		{
			for(int i = 0; i < im.rows*im.cols; i++)
			{
				depthData.push_back(10);
			}
			cutOff = 5;
		}
	}

	/*
	//view image read in for debugging
	vector<uint8_t> depthImg;
	for (int i = 0; i < depthData.size(); i++)
	{
		depthImg.push_back(255 - (depthData[i] * 255 / 300));
	}

	cv::Mat depthMat(600, 800, CV_8U, &depthImg[0]);
	cv::imshow("test", depthMat);
	cv::waitKey(0);
	std::cout << "got " << depthData.size() << "depth value" << std::endl;
	*/

	/**
	 * CALL THE CUDA INTERFACE FUNC
	 */
	int start_s = clock();
	cv::Mat output_mat = run_interpolation(im, depthData, cutOff, COLOURS_RAW);

	std::vector<cv::Mat> bgr;
	cv::split(output_mat, bgr);
	auto veg = cv::Scalar(156,41,156);

	cv::Mat mask = (bgr[0] == veg[0]) & (bgr[1] == veg[1]) & (bgr[2] == veg[2]);

	cv::morphologyEx(mask, mask, MORPH_CLOSE, kernel);
	output_mat.setTo(veg, mask);
	int stop_s= clock();
	
	/** ** **/
	fs::create_directories(fs::path(output_path).parent_path());
	std::vector<unsigned char> buf;
	cv::imencode(fs::path(output_path).extension().string(), output_mat, buf);
	//cv::imwrite(output_path + "t", output_mat);
	{
		std::ofstream ofs(output_path + "t", std::ofstream::binary);
		ofs.write((const char*)&buf[0], buf.size());
	}
	fs::rename(output_path + "t", output_path);

	// display execution time of the kernel function
	cout << "done in " << (stop_s-start_s)/double(CLOCKS_PER_SEC)*1000 << "msec!" << endl;
}

void processLoop(std::vector<std::string> toProcess, std::string image_path, std::string output_path, int device, cv::Mat kernel)
{
	gpuErrchk( hipSetDevice(device) );

	for(int i = 0; i < toProcess.size(); i++)
	{
		std::string curPath = toProcess[i];
		std::string outImgPath = std::regex_replace(curPath, std::regex(image_path), output_path);
		processImage(curPath, outImgPath, "fake", 20, kernel);
	}
}

template<typename T>
std::vector<std::vector<T>> SplitVector(const std::vector<T>& vec, size_t n)
{
    std::vector<std::vector<T>> outVec;

    size_t length = vec.size() / n;
    size_t remain = vec.size() % n;

    size_t begin = 0;
    size_t end = 0;

    for (size_t i = 0; i < std::min(n, vec.size()); ++i)
    {
        end += (remain > 0) ? (length + !!(remain--)) : length;

        outVec.push_back(std::vector<T>(vec.begin() + begin, vec.begin() + end));

        begin = end;
    }

    return outVec;
}

std::vector<int> parseDeviceList(std::string devList)
{
	std::vector<int> ret;
	std::vector<std::string> splitStr = split(devList, ',');
	for(std::string& s : splitStr)
	{
		ret.push_back(atoi(s.c_str()));
	}
	return ret;
}

/**
 * contains cuda specific initializations
 */
int main( int argc, char** argv )
{
	// grab the arguments
	string image_path, output_path, depth_path;
	int kernel_size = 7;
	int numProc = 8;
	std::vector<int> availDevice = {2}; //static max number because this uses a lot of GPU, so only one per GPU
	for (int i = 0; i < argc; i++)
	{
		if (strcmp(argv[i], "-i") == 0)
			image_path = argv[i+1];
		if (strcmp(argv[i], "-o") == 0)
			output_path = argv[i+1];
		if (strcmp(argv[i], "-d") == 0)
			depth_path = argv[i + 1];
		if (strcmp(argv[i], "-k") == 0)
			kernel_size = atoi(argv[i+1]);
		if (strcmp(argv[i], "-n") == 0)
			numProc = atoi(argv[i+1]);
		if (strcmp(argv[i], "-g") == 0)
			availDevice =  parseDeviceList(argv[i+1]);
	}

	std::vector<std::string> toProcess = GetImagesToProcess(image_path, output_path);
	if(toProcess.size() > 20 && numProc > 1)
	{
		std::cout << "over" << std::endl;
		std::vector<std::thread> threads;
		std::vector<std::vector<std::string>> splitVals = SplitVector(toProcess, numProc);
		for(int i = 0; i < numProc && i < splitVals.size(); i++)
		{
			int d = availDevice[i % availDevice.size()];
			threads.push_back(std::thread(processLoop, splitVals[i], image_path, output_path, d, kernel));
		}

		for(std::thread& t : threads)
		{
			t.join();
		}
	}
	else if(toProcess.size() > 0)
	{
		processLoop(toProcess, image_path, output_path, availDevice[0], kernel);
	}
}
