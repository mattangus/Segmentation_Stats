#include <vector>
#include <hipDNN.h>
#include <hip/hip_runtime.h>

#include "helpers.cuh"
#include "testHelper.cuh"
#include "tensorTests.cuh"
#include "common.cuh"

int main(int argc, char **argv) {
    std::cout << "cudnn ver: " << CUDNN_MAJOR << "." << CUDNN_MINOR << "." << CUDNN_PATCHLEVEL << std::endl;
    
    globalSetup();

    tensorTests::runAllTests();

    return 0;
}