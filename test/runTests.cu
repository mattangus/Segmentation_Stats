#include <vector>
#include <hipDNN.h>
#include <hip/hip_runtime.h>

#include "testHelper.cuh"
#include "tensorTests.cuh"

int main(int argc, char **argv) {
    std::cout << "cudnn ver: " << CUDNN_MAJOR << "." << CUDNN_MINOR << "." << CUDNN_PATCHLEVEL << std::endl;

    globalSetup();

    tensorTests::runAllTests();

    return 0;
}